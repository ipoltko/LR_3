#include "hip/hip_runtime.h"
// Copyright (c) 2017, Geometric Computation Group of Stanford University

// The MIT License (MIT)

// Copyright (c) 2017 Charles R. Qi

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_util.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// input: new_xyz(b, m, 3) xyz(b, n, 3)
// output: idx(b, m, nsample)
__global__ void query_ball_point_kernel(int b, int n, int m, float radius,
                                        int nsample,
                                        const float *__restrict__ new_xyz,
                                        const float *__restrict__ xyz,
                                        int *__restrict__ idx) {
  int batch_index = blockIdx.x;
  xyz += batch_index * n * 3;
  new_xyz += batch_index * m * 3;
  idx += m * nsample * batch_index;

  int index = threadIdx.x;
  int stride = blockDim.x;

  float radius2 = radius * radius;
  for (int j = index; j < m; j += stride) {
    float new_x = new_xyz[j * 3 + 0];
    float new_y = new_xyz[j * 3 + 1];
    float new_z = new_xyz[j * 3 + 2];
    for (int k = 0, cnt = 0; k < n && cnt < nsample; ++k) {
      float x = xyz[k * 3 + 0];
      float y = xyz[k * 3 + 1];
      float z = xyz[k * 3 + 2];
      float d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y) +
                 (new_z - z) * (new_z - z);
      if (d2 < radius2) {
        if (cnt == 0) {
          for (int l = 0; l < nsample; ++l) {
            idx[j * nsample + l] = k;
          }
        }
        idx[j * nsample + cnt] = k;
        ++cnt;
      }
    }
  }
}

void query_ball_point_kernel_launcher(int b, int n, int m, float radius,
                                      int nsample, const float *new_xyz,
                                      const float *xyz, int *idx) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  query_ball_point_kernel<<<b, opt_n_threads(m), 0, stream>>>(
      b, n, m, radius, nsample, new_xyz, xyz, idx);

  CUDA_CHECK_ERRORS();
}

// input: new_xyz(b, m, 3) xyz(b, n, 3)
// output: idx(b, m, nsample)
__global__ void query_ball_random_point_kernel(
    int seed, hiprandState *rand_states, int b, int n, int m, float radius,
    int nsample, const float *__restrict__ new_xyz,
    const float *__restrict__ xyz, int *__restrict__ idx) {

  int id = threadIdx.x + blockIdx.x * blockDim.x;
  hiprandState *local_state = rand_states + id;

  // TODO: optimize: hiprand_init is slow.
  hiprand_init(seed, id, 0, local_state);
  // // A potentially faster but less accurate version:
  // hiprand_init(seed + id * 1337, 0, 0, &rand_states[id]);

  int batch_index = blockIdx.x;
  xyz += batch_index * n * 3;
  new_xyz += batch_index * m * 3;
  idx += m * nsample * batch_index;

  int index = threadIdx.x;
  int stride = blockDim.x;

  float radius2 = radius * radius;
  for (int j = index; j < m; j += stride) {
    float new_x = new_xyz[j * 3 + 0];
    float new_y = new_xyz[j * 3 + 1];
    float new_z = new_xyz[j * 3 + 2];
    for (int k = 0, cnt = 0; k < n; ++k) {
      float x = xyz[k * 3 + 0];
      float y = xyz[k * 3 + 1];
      float z = xyz[k * 3 + 2];
      float d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y) +
                 (new_z - z) * (new_z - z);
      if (d2 < radius2) {
        if (cnt == 0) {
          for (int l = 0; l < nsample; ++l) {
            idx[j * nsample + l] = k;
          }
        } else if (cnt < nsample) {
          idx[j * nsample + cnt] = k;
        } else {
          unsigned int r = hiprand_uniform(local_state) * (cnt + 1);
          if (r < nsample) {
            idx[j * nsample + r] = k;
          }
        }
        ++cnt;
      }
    }
  }
}

void query_ball_random_point_kernel_launcher(int seed, int b, int n, int m,
                                             float radius, int nsample,
                                             const float *new_xyz,
                                             const float *xyz, int *idx) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int grid_dim = b;
  int block_dim = opt_n_threads(m);
  int num_threads = grid_dim * block_dim;

  hiprandState *rand_states;
  hipMalloc((void **)&rand_states, num_threads * sizeof(hiprandState));

  query_ball_random_point_kernel<<<grid_dim, block_dim, 0, stream>>>(
      seed, rand_states, b, n, m, radius, nsample, new_xyz, xyz, idx);

  hipFree(rand_states);

  CUDA_CHECK_ERRORS();
}

// input: points(b, c, n) idx(b, npoints, nsample)
// output: out(b, c, npoints, nsample)
__global__ void gather_by_index_kernel(int b, int c, int n, int npoints,
                                       int nsample,
                                       const float *__restrict__ points,
                                       const int *__restrict__ idx,
                                       float *__restrict__ out) {
  int batch_index = blockIdx.x;
  points += batch_index * n * c;
  idx += batch_index * npoints * nsample;
  out += batch_index * npoints * nsample * c;

  const int index = threadIdx.y * blockDim.x + threadIdx.x;
  const int stride = blockDim.y * blockDim.x;
  for (int i = index; i < c * npoints; i += stride) {
    const int l = i / npoints;
    const int j = i % npoints;
    for (int k = 0; k < nsample; ++k) {
      int ii = idx[j * nsample + k];
      out[(l * npoints + j) * nsample + k] = points[l * n + ii];
    }
  }
}

void gather_by_index_kernel_launcher(int b, int c, int n, int npoints,
                                     int nsample, const float *points,
                                     const int *idx, float *out) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  gather_by_index_kernel<<<b, opt_block_config(npoints, c), 0, stream>>>(
      b, c, n, npoints, nsample, points, idx, out);

  CUDA_CHECK_ERRORS();
}

// input: grad_out(b, c, npoints, nsample), idx(b, npoints, nsample)
// output: grad_points(b, c, n)
__global__ void gather_by_index_grad_kernel(int b, int c, int n, int npoints,
                                            int nsample,
                                            const float *__restrict__ grad_out,
                                            const int *__restrict__ idx,
                                            float *__restrict__ grad_points) {
  int batch_index = blockIdx.x;
  grad_out += batch_index * npoints * nsample * c;
  idx += batch_index * npoints * nsample;
  grad_points += batch_index * n * c;

  const int index = threadIdx.y * blockDim.x + threadIdx.x;
  const int stride = blockDim.y * blockDim.x;
  for (int i = index; i < c * npoints; i += stride) {
    const int l = i / npoints;
    const int j = i % npoints;
    for (int k = 0; k < nsample; ++k) {
      int ii = idx[j * nsample + k];
      atomicAdd(grad_points + l * n + ii,
                grad_out[(l * npoints + j) * nsample + k]);
    }
  }
}

void gather_by_index_grad_kernel_launcher(int b, int c, int n, int npoints,
                                          int nsample, const float *grad_out,
                                          const int *idx, float *grad_points) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  gather_by_index_grad_kernel<<<b, opt_block_config(npoints, c), 0, stream>>>(
      b, c, n, npoints, nsample, grad_out, idx, grad_points);

  CUDA_CHECK_ERRORS();
}
